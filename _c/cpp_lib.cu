#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <chrono>
#include <string>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include "sputnik/sputnik.h"
namespace py = pybind11;
#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            return EXIT_FAILURE;                                           \
        }                                                                  \
    }

// #define DEBUG

/**
 * @brief This function time the cuSparse sparse-dense matrix multiplication: A @ B where A is of
 * shape m * k and B is of shape k * n
 * @return float : the time in millisecond
 */

float test_cusparse_gemm(int m, int n, int k, int A_nnz, py::array_t<int> A_csr_offsets,
                         py::array_t<int> A_csr_columns, py::array_t<float> A_csr_values, py::array_t<float> arr_B)
{
    typedef std::chrono::steady_clock Clock;
    typedef std::chrono::nanoseconds nanoseconds;

    float alpha = 1.0f;
    float beta = 0.0f;
    int ldb = k;
    int ldc = m;

    //Get the array from the input
    py::buffer_info buf_A_csr_offsets = A_csr_offsets.request();
    py::buffer_info buf_A_csr_columns = A_csr_columns.request();
    py::buffer_info buf_A_csr_values = A_csr_values.request();
    py::buffer_info buf_B_values = arr_B.request();
    int *hA_csr_offsets = (int *)buf_A_csr_offsets.ptr;
    int *hA_csr_columns = (int *)buf_A_csr_columns.ptr;
    float *hA_csr_values = (float *)buf_A_csr_values.ptr;
    float *hB_values = (float *)buf_B_values.ptr;

    //device memory
    int *dA_csr_offsets, *dA_csr_columns;
    float *dA_csr_values, *dB_values, *dC_values;
    int A_num_rows = m;
    int A_num_cols = k;
    int B_num_rows = k;
    int B_num_cols = n;

    //allocate A
    CHECK_CUDA(hipMalloc((void **)&dA_csr_offsets, (A_num_rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&dA_csr_columns, A_nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&dA_csr_values, A_nnz * sizeof(float)));

    //allocate B
    CHECK_CUDA(hipMalloc((void **)&dB_values, sizeof(float) * B_num_rows * B_num_cols));
    //allocate C
    CHECK_CUDA(hipMalloc((void **)&dC_values, sizeof(float) * A_num_rows * B_num_cols));

    //to device mtx A
    CHECK_CUDA(hipMemcpy(dA_csr_offsets, hA_csr_offsets, (A_num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_csr_columns, hA_csr_columns, A_nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_csr_values, hA_csr_values, A_nnz * sizeof(float), hipMemcpyHostToDevice));
    //to device mtx B
    CHECK_CUDA(hipMemcpy(dB_values, hB_values, (B_num_rows * B_num_cols) * sizeof(float), hipMemcpyHostToDevice));
    //create the matrices
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                     dA_csr_offsets, dA_csr_columns, dA_csr_values, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, B_num_rows, B_num_cols, ldb, dB_values,
                                       HIP_R_32F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC_values,
                                       HIP_R_32F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    //SpGEMM
    Clock::time_point start = Clock::now();
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize))

    void *dBuffer = NULL;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    CHECK_CUSPARSE(hipsparseSpMM(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer))
    hipDeviceSynchronize();
    Clock::time_point end = Clock::now();
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC))
    CHECK_CUSPARSE(hipsparseDestroy(handle))

#ifdef DEBUG
    float *hC_values;
    hC_values = (float *)malloc(A_num_rows * B_num_cols * sizeof(float));
    //copy the result to the host
    CHECK_CUDA(hipMemcpy(hC_values, dC_values, A_num_rows * B_num_cols * sizeof(float), hipMemcpyDeviceToHost))
    //print out the result
    fprintf(stderr, "printing the multiplication result \n");
    for (int i = 0; i < A_num_rows * B_num_cols; i++)
    {
        fprintf(stderr, "%f\n", hC_values[i]);
    }
#endif
    //device memory free
    CHECK_CUDA(hipFree(dBuffer))
    CHECK_CUDA(hipFree(dA_csr_offsets))
    CHECK_CUDA(hipFree(dA_csr_columns))
    CHECK_CUDA(hipFree(dA_csr_values))
    CHECK_CUDA(hipFree(dB_values))
    CHECK_CUDA(hipFree(dC_values))
    //get the time
    nanoseconds ms = std::chrono::duration_cast<nanoseconds>(end - start);
    return ms.count();
}

float test_cublas_sgemm(int m, int n, int k, py::array_t<float> arr_A, py::array_t<float> arr_B)
{
    // float test_cublas_sgemm(int m, int n, int k, float * arr_A, float * arr_B) {
    //remember the mtx is col based!!!
    //init the variables
    typedef std::chrono::steady_clock Clock;
    typedef std::chrono::nanoseconds nanoseconds;
    float *A, *B;
    float *d_A, *d_B, *d_C;
#ifdef DEBUG
    //define the output variable C
    float *C;
    C = (float *)malloc(sizeof(float) * m * n);
#endif

    // get the elements inside the numpy passed in array
    py::buffer_info buf_A = arr_A.request();
    py::buffer_info buf_B = arr_B.request();
    A = (float *)buf_A.ptr;
    B = (float *)buf_B.ptr;

    // A = arr_A;
    // B = arr_B;

    //cuda code
    hipblasHandle_t handle;
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "fail handle");
    }

    CHECK_CUDA(hipMalloc((void **)&d_A, sizeof(float) * m * k))
    CHECK_CUDA(hipMalloc((void **)&d_B, sizeof(float) * n * k))
    CHECK_CUDA(hipMalloc((void **)&d_C, sizeof(float) * m * n))

    CHECK_CUDA(hipMemcpy(d_A, A, sizeof(float) * m * k, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(d_B, B, sizeof(float) * n * k, hipMemcpyHostToDevice))

    const float a = 1.0, b = 0.0;
    Clock::time_point start = Clock::now();

    hipblasStatus_t ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &a, d_A, m, d_B, k, &b, d_C, m);
    hipDeviceSynchronize();
    Clock::time_point end = Clock::now();

#ifdef DEBUG
    //copy the result back to host memory ofr latter printing
    CHECK_CUDA(hipMemcpy(C, d_C, sizeof(float) * m * n, hipMemcpyDeviceToHost))
#endif

    CHECK_CUDA(hipFree(d_A))
    CHECK_CUDA(hipFree(d_B))
    CHECK_CUDA(hipFree(d_C))

    hipblasDestroy(handle);

    nanoseconds ms = std::chrono::duration_cast<nanoseconds>(end - start);

#ifdef DEBUG
    fprintf(stderr, "printing the multiplication result col by col, matrix is %d X %d\n\n", m, n);
    for (int i = 0; i < m * n; i++)
    {
        fprintf(stderr, "%f \n", C[i]);
    }
#endif
    return ms.count();
}

float test_sgk_spmm(int m, int n, int k, int nonzeros, py::array_t<float> A_value, py::array_t<int> A_row_idex,
                    py::array_t<int> A_row_offsets, py::array_t<int> A_col_indices, py::array_t<float> B_values)
{

    typedef std::chrono::steady_clock Clock;
    typedef std::chrono::nanoseconds nanoseconds;

    //Get the array from the input
    py::buffer_info buf_A_value = A_value.request();
    py::buffer_info buf_A_row_idex = A_row_idex.request();
    py::buffer_info buf_A_row_offsets = A_row_offsets.request();
    py::buffer_info buf_A_col_indices = A_col_indices.request();
    py::buffer_info buf_B_values = B_values.request();

    float *h_values = (float *)buf_A_value.ptr;
    int *h_row_indices = (int *)buf_A_row_idex.ptr;
    int *h_row_offsets = (int *)buf_A_row_offsets.ptr;
    int *h_col_indices = (int *)buf_A_col_indices.ptr;
    float *h_dense_matrix = (float *)buf_B_values.ptr;

    int *row_indices, *row_offsets, *column_indices;
    float *values, *dense_matrix, *output_matrix;

    //allocate A
    CHECK_CUDA(hipMalloc((void **)&values, sizeof(float) * nonzeros))
    CHECK_CUDA(hipMalloc((void **)&row_indices, sizeof(int) * m))
    CHECK_CUDA(hipMalloc((void **)&row_offsets, sizeof(int) * (m + 1)))
    CHECK_CUDA(hipMalloc((void **)&column_indices, sizeof(int) * m * k))
    // allocate B
    CHECK_CUDA(hipMalloc((void **)&dense_matrix, sizeof(float) * n * k))
    // allocate C
    CHECK_CUDA(hipMalloc((void **)&output_matrix, sizeof(float) * n * m))

    //to device mtx A
    CHECK_CUDA(hipMemcpy(row_indices, h_row_indices, sizeof(int) * m, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(row_offsets, h_row_offsets, sizeof(int) * (m+1), hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(column_indices, h_col_indices, sizeof(int) * m * k, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(values, h_values, nonzeros * sizeof(float), hipMemcpyHostToDevice))
    //to device mtx B
    CHECK_CUDA(hipMemcpy(dense_matrix, h_dense_matrix, sizeof(float) * n * k, hipMemcpyHostToDevice))

    hipStream_t handle = NULL;
    CHECK_CUDA(hipStreamCreate(&handle))

    float* bias = nullptr;

    Clock::time_point start = Clock::now();
    CHECK_CUDA(sputnik::CudaSpmmBiasRelu(m, k, n, nonzeros, row_indices, values,
                                        row_offsets, column_indices, dense_matrix,
                                        bias, output_matrix, handle))
    hipDeviceSynchronize();
    Clock::time_point end = Clock::now();

    CHECK_CUDA(hipFree(row_indices))
    CHECK_CUDA(hipFree(row_offsets))
    CHECK_CUDA(hipFree(column_indices))
    CHECK_CUDA(hipFree(values))
    CHECK_CUDA(hipFree(dense_matrix))
    CHECK_CUDA(hipFree(output_matrix))

    CHECK_CUDA(hipStreamDestroy(handle))
    nanoseconds ms = std::chrono::duration_cast<nanoseconds>(end - start);
    return ms.count();
}

//Pybind call
PYBIND11_MODULE(cpp_lib, m)
{
    m.def("cuBLAS", &test_cublas_sgemm, "the function returning the RT of cuBLAS");
    m.def("cuSPARSE", &test_cusparse_gemm, "the function returning the RT of cuSPARSE");
    m.def("sgkSPARSE", &test_sgk_spmm, "the function returning the RT of sgk");
}